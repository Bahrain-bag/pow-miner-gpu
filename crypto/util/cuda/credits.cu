/* creditscoin SHA256 djm34 implementation - 2015 */

#include <iostream>

#include "miner.h"
#include "cuda_helper.h"
#include "td/utils/Slice-decl.h"
#include "td/utils/misc.h"

extern void bitcredit_setBlockTarget(uint32_t cpu_id, unsigned char *data, const void *ptarget, unsigned char *rdata);
extern void bitcredit_cpu_init(uint32_t gpu_id, uint32_t cpu_id, uint64_t threads);
extern HashResult bitcredit_cpu_hash(uint32_t gpu_id, uint32_t cpu_id, uint64_t threads, uint64_t startNounce, uint32_t expired);

static bool init = false;

extern "C" int scanhash_credits(int gpu_id, int cpu_id, ton::HDataEnv H, const ton::Miner::Options &options, uint64_t *pdata,
                                const uint32_t *ptarget, uint64_t max_nonce, unsigned char *rdata) {
  td::Slice data = H.as_slice();
  constexpr size_t prefix_size = 72;
  td::Slice head = data.substr(0, prefix_size);
  td::Slice tail = data.substr(prefix_size);
  char guard = head.back();

  // throughput
  td::uint64 throughput = device_intensity(gpu_id, __func__, 1U << 25); // 256*256*64*8
  if (options.max_iterations < throughput) {
    throughput = options.max_iterations;
  }
  std::cout << cpu_id << ": " << "GPU throughput: " << throughput << ", VCPUS: " << MAX_VCPUS << std::endl;

  // cuda device
  if (!init) {
    hipSetDevice(device_map[gpu_id]);
    hipSetDeviceFlags(hipDeviceScheduleBlockingSync);
    hipDeviceSetCacheConfig(hipFuncCachePreferL1);
    init = true;
  }

  // allocate mem
  bitcredit_cpu_init(gpu_id, cpu_id, throughput);

  // set data
  // std::cout << "data: " << hex_encode(data) << std::endl;
  unsigned char input[123], complexity[32];
  memcpy(input, data.ubegin(), data.size());
  bitcredit_setBlockTarget(cpu_id, input, options.complexity.data(), rdata);

  uint32_t expired;
  td::int64 i = 0;
  for (; i < options.max_iterations; i += throughput) {
    expired = (uint32_t)td::Clocks::system() + 900;
    HashResult foundNonce = bitcredit_cpu_hash(gpu_id, cpu_id, throughput, i, expired);
    if (foundNonce.nonce != UINT64_MAX) {
      pdata[0] = foundNonce.nonce;
      pdata[1] = foundNonce.vcpu;
      pdata[2] = expired;
      if (options.hashes_computed) {
        *options.hashes_computed += i + foundNonce.nonce * foundNonce.vcpu;
      }
      return 1;
    }
    if ((foundNonce.nonce + throughput) > UINT64_MAX) {
      pdata[0] = UINT64_MAX;
      pdata[1] = foundNonce.vcpu;
      pdata[2] = expired;
      if (options.hashes_computed) {
        *options.hashes_computed += i + UINT64_MAX * foundNonce.vcpu;
      }
      return 0;
    }
    if (options.token_) {
      break;
    }
    if (options.expire_at && options.expire_at.value().is_in_past(td::Timestamp::now())) {
      break;
    }
  }
  if (options.hashes_computed) {
    *options.hashes_computed += i;
  }
  return 0;
}
