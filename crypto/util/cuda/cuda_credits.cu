#include "hip/hip_runtime.h"
/* CreditsCoin [CRD] djm34 implementation - 2015 */

#include <stdio.h>
#include <memory.h>
#include <iostream>

#include "cuda_vector.h"
#include "miner.h"

uint64_t *d_BitNonce[MAX_CPUS];
uint64_t *d_BitVcpu[MAX_CPUS];
__constant__ uint8_t c_rdata[MAX_CPUS * 32 * MAX_GPU_THREADS];
__constant__ uint32_t pTarget[8];  // 8*4 = 32
__constant__ uint32_t c_data[48];  // 48*4 = 192

///////////////////////////////////////////////////////////////////////////////////
/////////////////////////////// sha256 Transform function /////////////////////////

static __constant__ uint8 H256 = {0x6A09E667, 0xBB67AE85, 0x3C6EF372, 0xA54FF53A,
                                  0x510E527F, 0x9B05688C, 0x1F83D9AB, 0x5BE0CD19};

static __constant__ uint32_t Ksha[64] = {
    0x428A2F98, 0x71374491, 0xB5C0FBCF, 0xE9B5DBA5, 0x3956C25B, 0x59F111F1, 0x923F82A4, 0xAB1C5ED5,
    0xD807AA98, 0x12835B01, 0x243185BE, 0x550C7DC3, 0x72BE5D74, 0x80DEB1FE, 0x9BDC06A7, 0xC19BF174,
    0xE49B69C1, 0xEFBE4786, 0x0FC19DC6, 0x240CA1CC, 0x2DE92C6F, 0x4A7484AA, 0x5CB0A9DC, 0x76F988DA,
    0x983E5152, 0xA831C66D, 0xB00327C8, 0xBF597FC7, 0xC6E00BF3, 0xD5A79147, 0x06CA6351, 0x14292967,
    0x27B70A85, 0x2E1B2138, 0x4D2C6DFC, 0x53380D13, 0x650A7354, 0x766A0ABB, 0x81C2C92E, 0x92722C85,
    0xA2BFE8A1, 0xA81A664B, 0xC24B8B70, 0xC76C51A3, 0xD192E819, 0xD6990624, 0xF40E3585, 0x106AA070,
    0x19A4C116, 0x1E376C08, 0x2748774C, 0x34B0BCB5, 0x391C0CB3, 0x4ED8AA4A, 0x5B9CCA4F, 0x682E6FF3,
    0x748F82EE, 0x78A5636F, 0x84C87814, 0x8CC70208, 0x90BEFFFA, 0xA4506CEB, 0xBEF9A3F7, 0xC67178F2};

static __device__ __forceinline__ uint32_t bsg2_0(const uint32_t x) {
  uint32_t r1 = ROTR32(x, 2);
  uint32_t r2 = ROTR32(x, 13);
  uint32_t r3 = ROTR32(x, 22);
  return xor3b(r1, r2, r3);
}

static __device__ __forceinline__ uint32_t bsg2_1(const uint32_t x) {
  uint32_t r1 = ROTR32(x, 6);
  uint32_t r2 = ROTR32(x, 11);
  uint32_t r3 = ROTR32(x, 25);
  return xor3b(r1, r2, r3);
}

static __device__ __forceinline__ uint32_t ssg2_0(const uint32_t x) {
  uint32_t r1 = ROTR32(x, 7);
  uint32_t r2 = ROTR32(x, 18);
  uint32_t r3 = shr_t32(x, 3);
  return xor3b(r1, r2, r3);
}

static __device__ __forceinline__ uint32_t ssg2_1(const uint32_t x) {
  uint32_t r1 = ROTR32(x, 17);
  uint32_t r2 = ROTR32(x, 19);
  uint32_t r3 = shr_t32(x, 10);
  return xor3b(r1, r2, r3);
}

static __device__ __forceinline__ void sha2_step1(const uint32_t a, const uint32_t b, const uint32_t c, uint32_t &d,
                                                  const uint32_t e, const uint32_t f, const uint32_t g, uint32_t &h,
                                                  const uint32_t in, const uint32_t Kshared) {
  uint32_t t1, t2;
  uint32_t vxandx = xandx(e, f, g);
  uint32_t bsg21 = bsg2_1(e);
  uint32_t bsg20 = bsg2_0(a);
  uint32_t andorv = andor32(a, b, c);

  t1 = h + bsg21 + vxandx + Kshared + in;
  t2 = bsg20 + andorv;
  d = d + t1;
  h = t1 + t2;
}

static __device__ __forceinline__ void sha2_step2(const uint32_t a, const uint32_t b, const uint32_t c, uint32_t &d,
                                                  const uint32_t e, const uint32_t f, const uint32_t g, uint32_t &h,
                                                  uint32_t *in, const uint32_t pc, const uint32_t Kshared) {
  uint32_t t1, t2;

  int pcidx1 = (pc - 2) & 0xF;
  int pcidx2 = (pc - 7) & 0xF;
  int pcidx3 = (pc - 15) & 0xF;
  uint32_t inx0 = in[pc];
  uint32_t inx1 = in[pcidx1];
  uint32_t inx2 = in[pcidx2];
  uint32_t inx3 = in[pcidx3];

  uint32_t ssg21 = ssg2_1(inx1);
  uint32_t ssg20 = ssg2_0(inx3);
  uint32_t vxandx = xandx(e, f, g);
  uint32_t bsg21 = bsg2_1(e);
  uint32_t bsg20 = bsg2_0(a);
  uint32_t andorv = andor32(a, b, c);

  in[pc] = ssg21 + inx2 + ssg20 + inx0;

  t1 = h + bsg21 + vxandx + Kshared + in[pc];
  t2 = bsg20 + andorv;
  d = d + t1;
  h = t1 + t2;
}

static __device__ __forceinline__ uint8 sha256_Transform2(uint16 in[1],
                                                          const uint8 &r)  // also known as sha2_round_body
{
  uint8 tmp = r;
#define a tmp.s0
#define b tmp.s1
#define c tmp.s2
#define d tmp.s3
#define e tmp.s4
#define f tmp.s5
#define g tmp.s6
#define h tmp.s7

  sha2_step1(a, b, c, d, e, f, g, h, in[0].s0, Ksha[0]);
  sha2_step1(h, a, b, c, d, e, f, g, in[0].s1, Ksha[1]);
  sha2_step1(g, h, a, b, c, d, e, f, in[0].s2, Ksha[2]);
  sha2_step1(f, g, h, a, b, c, d, e, in[0].s3, Ksha[3]);
  sha2_step1(e, f, g, h, a, b, c, d, in[0].s4, Ksha[4]);
  sha2_step1(d, e, f, g, h, a, b, c, in[0].s5, Ksha[5]);
  sha2_step1(c, d, e, f, g, h, a, b, in[0].s6, Ksha[6]);
  sha2_step1(b, c, d, e, f, g, h, a, in[0].s7, Ksha[7]);
  sha2_step1(a, b, c, d, e, f, g, h, in[0].s8, Ksha[8]);
  sha2_step1(h, a, b, c, d, e, f, g, in[0].s9, Ksha[9]);
  sha2_step1(g, h, a, b, c, d, e, f, in[0].sa, Ksha[10]);
  sha2_step1(f, g, h, a, b, c, d, e, in[0].sb, Ksha[11]);
  sha2_step1(e, f, g, h, a, b, c, d, in[0].sc, Ksha[12]);
  sha2_step1(d, e, f, g, h, a, b, c, in[0].sd, Ksha[13]);
  sha2_step1(c, d, e, f, g, h, a, b, in[0].se, Ksha[14]);
  sha2_step1(b, c, d, e, f, g, h, a, in[0].sf, Ksha[15]);

#pragma unroll
  for (int i = 0; i < 3; i++) {
    sha2_step2(a, b, c, d, e, f, g, h, (uint32_t *)in, 0, Ksha[16 + 16 * i]);
    sha2_step2(h, a, b, c, d, e, f, g, (uint32_t *)in, 1, Ksha[17 + 16 * i]);
    sha2_step2(g, h, a, b, c, d, e, f, (uint32_t *)in, 2, Ksha[18 + 16 * i]);
    sha2_step2(f, g, h, a, b, c, d, e, (uint32_t *)in, 3, Ksha[19 + 16 * i]);
    sha2_step2(e, f, g, h, a, b, c, d, (uint32_t *)in, 4, Ksha[20 + 16 * i]);
    sha2_step2(d, e, f, g, h, a, b, c, (uint32_t *)in, 5, Ksha[21 + 16 * i]);
    sha2_step2(c, d, e, f, g, h, a, b, (uint32_t *)in, 6, Ksha[22 + 16 * i]);
    sha2_step2(b, c, d, e, f, g, h, a, (uint32_t *)in, 7, Ksha[23 + 16 * i]);
    sha2_step2(a, b, c, d, e, f, g, h, (uint32_t *)in, 8, Ksha[24 + 16 * i]);
    sha2_step2(h, a, b, c, d, e, f, g, (uint32_t *)in, 9, Ksha[25 + 16 * i]);
    sha2_step2(g, h, a, b, c, d, e, f, (uint32_t *)in, 10, Ksha[26 + 16 * i]);
    sha2_step2(f, g, h, a, b, c, d, e, (uint32_t *)in, 11, Ksha[27 + 16 * i]);
    sha2_step2(e, f, g, h, a, b, c, d, (uint32_t *)in, 12, Ksha[28 + 16 * i]);
    sha2_step2(d, e, f, g, h, a, b, c, (uint32_t *)in, 13, Ksha[29 + 16 * i]);
    sha2_step2(c, d, e, f, g, h, a, b, (uint32_t *)in, 14, Ksha[30 + 16 * i]);
    sha2_step2(b, c, d, e, f, g, h, a, (uint32_t *)in, 15, Ksha[31 + 16 * i]);
  }

#undef a
#undef b
#undef c
#undef d
#undef e
#undef f
  return (r + tmp);
}

__global__ __launch_bounds__(256, 3) void bitcredit_gpu_hash(uint32_t gpu_threads, uint32_t cpu_id, uint64_t threads,
                                                             uint64_t startNonce, uint32_t expired,
                                                             uint64_t *NonceVector, uint64_t *VcpuVector) {
  // 2d grid of 1d blocks
  // int blockId = blockIdx.y * gridDim.x + blockIdx.x;
  // int threadId = blockId * blockDim.x + threadIdx.x;
  int vcpu = blockIdx.y;
  uint64_t thread = blockIdx.x * blockDim.x + threadIdx.x;
  uint64_t nonce = startNonce + thread;

  uint16 data[1];
  uint8 state = H256;

  // PaddedMessage[192]:
  // 00f24d69 6e650000 000000aa aaaaaaaa aaaaaaaa aaaaaaaa aaaaaaaa aaaaaaaa aaaaaaaa aaaaaaaa aaaaaa00 00000000 00000000 00000000 00000000 00000000
  // 00000000 00000000 000000e6 40a697b2 9adcc54c 26404abe 70352f00 00000000 00000000 00000000 00000000 00000000 00000000 00000000 00000080 00000000
  // 00000000 00000000 00000000 00000000 00000000 00000000 00000000 00000000 00000000 00000000 00000000 00000000 00000000 00000000 00000000 000003d8

  // read rdata from offset
  // 24:31 bytes
  uint64_t rdata1 = ((uint64_t)c_rdata[cpu_id * 32 * gpu_threads + 32 * vcpu + 24]) << 56 |
                    ((uint64_t)c_rdata[cpu_id * 32 * gpu_threads + 32 * vcpu + 25]) << 48 |
                    ((uint64_t)c_rdata[cpu_id * 32 * gpu_threads + 32 * vcpu + 26]) << 40 |
                    ((uint64_t)c_rdata[cpu_id * 32 * gpu_threads + 32 * vcpu + 27]) << 32 |
                    ((uint64_t)c_rdata[cpu_id * 32 * gpu_threads + 32 * vcpu + 28]) << 24 |
                    ((uint64_t)c_rdata[cpu_id * 32 * gpu_threads + 32 * vcpu + 29]) << 16 |
                    ((uint64_t)c_rdata[cpu_id * 32 * gpu_threads + 32 * vcpu + 30]) << 8 |
                    ((uint64_t)c_rdata[cpu_id * 32 * gpu_threads + 32 * vcpu + 31]);

  // increment rdata1 & rdata2
  rdata1 += nonce;

  uint32_t rdata01 = (c_data[1] & ~(0xff)) | (uint8_t)(expired >> 24);
  uint32_t rdata02 = ((expired << 8) & ~(0xff)) | (uint8_t)(c_data[2]);

  uint32_t rdata10 = (c_data[10] & ~(0xff)) | (uint8_t)c_rdata[cpu_id * 32 * gpu_threads + 32 * vcpu + 0];
  uint32_t rdata11 = (uint32_t)c_rdata[cpu_id * 32 * gpu_threads + 32 * vcpu + 1] << 24 |
                     (uint32_t)c_rdata[cpu_id * 32 * gpu_threads + 32 * vcpu + 2] << 16 |
                     (uint32_t)c_rdata[cpu_id * 32 * gpu_threads + 32 * vcpu + 3] << 8 |
                     (uint32_t)c_rdata[cpu_id * 32 * gpu_threads + 32 * vcpu + 4];
  uint32_t rdata12 = (uint32_t)c_rdata[cpu_id * 32 * gpu_threads + 32 * vcpu + 5] << 24 |
                     (uint32_t)c_rdata[cpu_id * 32 * gpu_threads + 32 * vcpu + 6] << 16 |
                     (uint32_t)c_rdata[cpu_id * 32 * gpu_threads + 32 * vcpu + 7] << 8 |
                     (uint32_t)c_rdata[cpu_id * 32 * gpu_threads + 32 * vcpu + 8];
  uint32_t rdata13 = (uint32_t)c_rdata[cpu_id * 32 * gpu_threads + 32 * vcpu + 9] << 24 |
                     (uint32_t)c_rdata[cpu_id * 32 * gpu_threads + 32 * vcpu + 10] << 16 |
                     (uint32_t)c_rdata[cpu_id * 32 * gpu_threads + 32 * vcpu + 11] << 8 |
                     (uint32_t)c_rdata[cpu_id * 32 * gpu_threads + 32 * vcpu + 12];
  uint32_t rdata14 = (uint32_t)c_rdata[cpu_id * 32 * gpu_threads + 32 * vcpu + 13] << 24 |
                     (uint32_t)c_rdata[cpu_id * 32 * gpu_threads + 32 * vcpu + 14] << 16 |
                     (uint32_t)c_rdata[cpu_id * 32 * gpu_threads + 32 * vcpu + 15] << 8 |
                     (uint32_t)c_rdata[cpu_id * 32 * gpu_threads + 32 * vcpu + 16];
  uint32_t rdata15 = (uint32_t)c_rdata[cpu_id * 32 * gpu_threads + 32 * vcpu + 17] << 24 |
                     (uint32_t)c_rdata[cpu_id * 32 * gpu_threads + 32 * vcpu + 18] << 16 |
                     (uint32_t)c_rdata[cpu_id * 32 * gpu_threads + 32 * vcpu + 19] << 8 |
                     (uint32_t)c_rdata[cpu_id * 32 * gpu_threads + 32 * vcpu + 20];
  uint32_t rdata16 = (uint32_t)c_rdata[cpu_id * 32 * gpu_threads + 32 * vcpu + 21] << 24 |
                     (uint32_t)c_rdata[cpu_id * 32 * gpu_threads + 32 * vcpu + 22] << 16 |
                     (uint32_t)c_rdata[cpu_id * 32 * gpu_threads + 32 * vcpu + 23] << 8 | (uint8_t)(rdata1 >> 56);
  uint32_t rdata17 = (uint32_t)(rdata1 >> 24);
  uint32_t rdata18 = ((uint32_t)(rdata1 << 8) & ~(0xff)) | (uint8_t)(c_data[18]);

  uint32_t rdata22 = (c_data[22] & ~(0xff)) | (uint8_t)c_rdata[cpu_id * 32 * gpu_threads + 32 * vcpu + 0];
  uint32_t rdata23 = rdata11;
  uint32_t rdata24 = rdata12;
  uint32_t rdata25 = rdata13;
  uint32_t rdata26 = rdata14;
  uint32_t rdata27 = rdata15;
  uint32_t rdata28 = rdata16;
  uint32_t rdata29 = rdata17;
  uint32_t rdata30 = ((uint32_t)(rdata1 << 8) & ~(0xff)) | (uint8_t)(c_data[30]);
  //  printf(
  //      "[%d:%lld]: rdata1=%016llX rdata01=%04x rdata02=%04x rdata16=%04x rdata17=%04x rdata18=%04x rdata28=%04x "
  //      "rdata29=%04x rdata30=%04x\n",
  //      vcpu, nonce, rdata1, rdata01, rdata02, rdata16, rdata17, rdata18, rdata28, rdata29, rdata30);

  // first block
  data[0].s0 = c_data[0];
  data[0].s1 = rdata01;  //c_data[1];
  data[0].s2 = rdata02;  //c_data[2];
  data[0].s3 = c_data[3];
  data[0].s4 = c_data[4];
  data[0].s5 = c_data[5];
  data[0].s6 = c_data[6];
  data[0].s7 = c_data[7];
  data[0].s8 = c_data[8];
  data[0].s9 = c_data[9];
  data[0].sa = rdata10;  //c_data[10];
  data[0].sb = rdata11;  //c_data[11];
  data[0].sc = rdata12;  //c_data[12];
  data[0].sd = rdata13;  //c_data[13];
  data[0].se = rdata14;  //c_data[14];
  data[0].sf = rdata15;  //c_data[15];
  //  printf("data0[%d:%lld]: %04x %04x %04x %04x %04x %04x %04x %04x %04x %04x %04x %04x %04x %04x %04x %04x\n", vcpu,
  //         nonce, c_data[0], rdata01, rdata02, c_data[3], c_data[4], c_data[5], c_data[6], c_data[7], c_data[8],
  //         c_data[9], rdata10, rdata11, rdata12, rdata13, rdata14, rdata15);
  state = sha256_Transform2(data, state);

  // second block
  data[0].s0 = rdata16;  //c_data[16];
  data[0].s1 = rdata17;  //c_data[17];
  data[0].s2 = rdata18;  //c_data[18];
  data[0].s3 = c_data[19];
  data[0].s4 = c_data[20];
  data[0].s5 = c_data[21];
  data[0].s6 = rdata22;  //c_data[22];
  data[0].s7 = rdata23;  //c_data[23];
  data[0].s8 = rdata24;  //c_data[24];
  data[0].s9 = rdata25;  //c_data[25];
  data[0].sa = rdata26;  //c_data[26];
  data[0].sb = rdata27;  //c_data[27];
  data[0].sc = rdata28;  //c_data[28];
  data[0].sd = rdata29;  //c_data[29];
  data[0].se = rdata30;  //c_data[30];
  data[0].sf = c_data[31];
  //  printf("data1[%d:%lld]: %04x %04x %04x %04x %04x %04x %04x %04x %04x %04x %04x %04x %04x %04x %04x %04x\n", vcpu,
  //         nonce, rdata16, rdata17, rdata18, c_data[19], c_data[20], c_data[21], rdata22, rdata23, rdata24, rdata25,
  //         rdata26, rdata27, rdata28, rdata29, rdata30, c_data[31]);
  state = sha256_Transform2(data, state);

  // third block
  data[0].s0 = c_data[32];
  data[0].s1 = c_data[33];
  data[0].s2 = c_data[34];
  data[0].s3 = c_data[35];
  data[0].s4 = c_data[36];
  data[0].s5 = c_data[37];
  data[0].s6 = c_data[38];
  data[0].s7 = c_data[39];
  data[0].s8 = c_data[40];
  data[0].s9 = c_data[41];
  data[0].sa = c_data[42];
  data[0].sb = c_data[43];
  data[0].sc = c_data[44];
  data[0].sd = c_data[45];
  data[0].se = c_data[46];
  data[0].sf = c_data[47];
  //  printf("data2[%d:%lld]: %04x %04x %04x %04x %04x %04x %04x %04x %04x %04x %04x %04x %04x %04x %04x %04x\n", vcpu, nonce,
  //         c_data[32], c_data[33], c_data[34], c_data[35], c_data[36], c_data[37], c_data[38], c_data[39], c_data[40],
  //         c_data[41], c_data[42], c_data[43], c_data[44], c_data[45], c_data[46], c_data[47]);
  state = sha256_Transform2(data, state);

  //  printf("hash[%lld]: %04x %04x %04x %04x %04x %04x %04x %04x\n", nonce, state.s0, state.s1, state.s2, state.s3,
  //         state.s4, state.s5, state.s6, state.s7);

  //  printf("complexity[%lld]: %04x %04x %04x %04x %04x %04x %04x %04x\n", nonce, pTarget[0], pTarget[1], pTarget[2],
  //         pTarget[3], pTarget[4], pTarget[5], pTarget[6], pTarget[7]);

  // alternative memcmp
  // check s0
  if (state.s0 < pTarget[0]) {
    NonceVector[0] = nonce;
    VcpuVector[0] = vcpu;
  } else if (state.s0 > pTarget[0])
    return;
  else if (state.s0 == pTarget[0]) {
    // check s1
    if (state.s1 < pTarget[1]) {
      //      printf("%d: hash[%d:%lld]: %04x = %04x, %04x < %04x\n", cpu_id, vcpu, nonce, state.s0, pTarget[0], state.s1,
      //             pTarget[1]);
      //
      //      printf("%d: data0[%d:%lld]: %04x %04x %04x %04x %04x %04x %04x %04x %04x %04x %04x %04x %04x %04x %04x %04x\n",
      //             cpu_id, vcpu, nonce, c_data[0], rdata01, rdata02, c_data[3], c_data[4], c_data[5], c_data[6], c_data[7],
      //             c_data[8], c_data[9], rdata10, rdata11, rdata12, rdata13, rdata14, rdata15);
      //
      //      printf("%d: data1[%d:%lld]: %04x %04x %04x %04x %04x %04x %04x %04x %04x %04x %04x %04x %04x %04x %04x %04x\n",
      //             cpu_id, vcpu, nonce, rdata16, rdata17, rdata18, c_data[19], c_data[20], c_data[21], rdata22, rdata23,
      //             rdata24, rdata25, rdata26, rdata27, rdata28, rdata29, rdata30, c_data[31]);
      //
      //      printf("%d: data2[%d:%lld]: %04x %04x %04x %04x %04x %04x %04x %04x %04x %04x %04x %04x %04x %04x %04x %04x\n",
      //             cpu_id, vcpu, nonce, c_data[32], c_data[33], c_data[34], c_data[35], c_data[36], c_data[37], c_data[38],
      //             c_data[39], c_data[40], c_data[41], c_data[42], c_data[43], c_data[44], c_data[45], c_data[46],
      //             c_data[47]);
      //
      //      printf("%d: hash[%d:%lld]: %04x %04x %04x %04x %04x %04x %04x %04x\n", cpu_id, vcpu, nonce, state.s0, state.s1,
      //             state.s2, state.s3, state.s4, state.s5, state.s6, state.s7);
      //
      //      printf("%d: complexity[%d:%lld]: %04x %04x %04x %04x %04x %04x %04x %04x\n", cpu_id, vcpu, nonce, pTarget[0],
      //             pTarget[1], pTarget[2], pTarget[3], pTarget[4], pTarget[5], pTarget[6], pTarget[7]);
      //
      //      printf("%d: rdata[%d:%lld]: %02x%02x%02x%02x %02x%02x%02x%02x %02x%02x%02x%02x %02x%02x%02x%02x", cpu_id, vcpu,
      //             nonce, c_rdata[cpu_id * 32 * gpu_threads + 32 * vcpu + 0], c_rdata[cpu_id * 32 * gpu_threads + 32 * vcpu + 1],
      //             c_rdata[cpu_id * 32 * gpu_threads + 32 * vcpu + 2], c_rdata[cpu_id * 32 * gpu_threads + 32 * vcpu + 3],
      //             c_rdata[cpu_id * 32 * gpu_threads + 32 * vcpu + 4], c_rdata[cpu_id * 32 * gpu_threads + 32 * vcpu + 5],
      //             c_rdata[cpu_id * 32 * gpu_threads + 32 * vcpu + 6], c_rdata[cpu_id * 32 * gpu_threads + 32 * vcpu + 7],
      //             c_rdata[cpu_id * 32 * gpu_threads + 32 * vcpu + 8], c_rdata[cpu_id * 32 * gpu_threads + 32 * vcpu + 9],
      //             c_rdata[cpu_id * 32 * gpu_threads + 32 * vcpu + 10], c_rdata[cpu_id * 32 * gpu_threads + 32 * vcpu + 11],
      //             c_rdata[cpu_id * 32 * gpu_threads + 32 * vcpu + 12], c_rdata[cpu_id * 32 * gpu_threads + 32 * vcpu + 13],
      //             c_rdata[cpu_id * 32 * gpu_threads + 32 * vcpu + 14], c_rdata[cpu_id * 32 * gpu_threads + 32 * vcpu + 15]);
      //
      //      printf("%02x%02x%02x%02x %02x%02x%02x%02x %02x%02x%02x%02x %02x%02x%02x%02x\n",
      //             c_rdata[cpu_id * 32 * gpu_threads + 32 * vcpu + 16], c_rdata[cpu_id * 32 * gpu_threads + 32 * vcpu + 17],
      //             c_rdata[cpu_id * 32 * gpu_threads + 32 * vcpu + 18], c_rdata[cpu_id * 32 * gpu_threads + 32 * vcpu + 19],
      //             c_rdata[cpu_id * 32 * gpu_threads + 32 * vcpu + 20], c_rdata[cpu_id * 32 * gpu_threads + 32 * vcpu + 21],
      //             c_rdata[cpu_id * 32 * gpu_threads + 32 * vcpu + 22], c_rdata[cpu_id * 32 * gpu_threads + 32 * vcpu + 23],
      //             c_rdata[cpu_id * 32 * gpu_threads + 32 * vcpu + 24], c_rdata[cpu_id * 32 * gpu_threads + 32 * vcpu + 25],
      //             c_rdata[cpu_id * 32 * gpu_threads + 32 * vcpu + 26], c_rdata[cpu_id * 32 * gpu_threads + 32 * vcpu + 27],
      //             c_rdata[cpu_id * 32 * gpu_threads + 32 * vcpu + 28], c_rdata[cpu_id * 32 * gpu_threads + 32 * vcpu + 29],
      //             c_rdata[cpu_id * 32 * gpu_threads + 32 * vcpu + 30], c_rdata[cpu_id * 32 * gpu_threads + 32 * vcpu + 31]);

      NonceVector[0] = nonce;
      VcpuVector[0] = vcpu;
    } else if (state.s1 > pTarget[1]) {
      return;
    } else if (state.s1 == pTarget[1]) {
      // check s2
      if (state.s2 < pTarget[2]) {
        NonceVector[0] = nonce;
        VcpuVector[0] = vcpu;
      } else if (state.s2 > pTarget[2]) {
        return;
      } else if (state.s2 == pTarget[2]) {
        // check s3
        if (state.s3 < pTarget[3]) {
          NonceVector[0] = nonce;
          VcpuVector[0] = vcpu;
        } else if (state.s3 > pTarget[3]) {
          return;
        } else if (state.s3 == pTarget[3]) {
          // check s4
          if (state.s4 < pTarget[4]) {
            NonceVector[0] = nonce;
            VcpuVector[0] = vcpu;
          } else if (state.s4 > pTarget[4]) {
            return;
          } else if (state.s4 == pTarget[4]) {
            // check s5
            if (state.s5 < pTarget[5]) {
              NonceVector[0] = nonce;
              VcpuVector[0] = vcpu;
            } else if (state.s5 > pTarget[5]) {
              return;
            } else if (state.s5 == pTarget[5]) {
              // check s6
              if (state.s6 < pTarget[6]) {
                NonceVector[0] = nonce;
                VcpuVector[0] = vcpu;
              } else if (state.s6 > pTarget[6]) {
                return;
              } else if (state.s6 == pTarget[6]) {
                // check s7
                if (state.s7 < pTarget[7]) {
                  NonceVector[0] = nonce;
                  VcpuVector[0] = vcpu;
                } else {
                  return;
                }
              }
            }
          }
        }
      }
    }
  }
}

__host__ void bitcredit_cpu_init(uint32_t gpu_id, uint32_t cpu_id, uint64_t threads) {
  hipMalloc(&d_BitNonce[cpu_id], sizeof(uint64_t));
  hipMalloc(&d_BitVcpu[cpu_id], sizeof(uint64_t));
}

__host__ bool bitcredit_setBlockTarget(uint32_t gpu_id, uint32_t gpu_threads, uint32_t cpu_id, unsigned char *data, const void *target,
                                       unsigned char *rdata) {
  
#ifndef _WIN32
  int len = 123, n = 3;
#else
  int len = 123;
  const int n = 3;
#endif
  uint32_t PaddedMessage[16 * n];  // bring balance to the force, 512*3 bits
  memset(PaddedMessage, 0, 16 * n * sizeof(uint32_t));
  memcpy(PaddedMessage, data, len);
  ((uchar *)PaddedMessage)[len] = 0x80;  // guard bit after data
  uint32_t endiandata[16 * n];
  for (int k = 0; k < 16 * n; k++)
    be32enc(&endiandata[k], ((uint32_t *)PaddedMessage)[k]);
  ((uint32_t *)endiandata)[16 * n - 1] = len * 8;  // size to the end

  uint32_t endiantarget[8];
  for (int k = 0; k < 8; k++)
    be32enc(&endiantarget[k], ((uint32_t *)target)[k]);

  //    std::cout << "PaddedMessage[" << 16 * n * sizeof(uint32_t) << "]: ";
  //    for (int z = 0; z < 16 * n; z++)
  //      printf("%08x ", endiandata[z]);
  //    std::cout << std::endl;

  CUDA_CALL_OR_RET_X(hipMemcpyToSymbol(HIP_SYMBOL(pTarget), endiantarget, 8 * sizeof(uint32_t), 0, hipMemcpyHostToDevice), false);
  CUDA_CALL_OR_RET_X(hipMemcpyToSymbol(HIP_SYMBOL(c_data), endiandata, 16 * n * sizeof(uint32_t), 0, hipMemcpyHostToDevice),
                     false);
  CUDA_CALL_OR_RET_X(hipMemcpyToSymbol(HIP_SYMBOL(c_rdata), rdata, 32 * gpu_threads * sizeof(uint8_t),
                                        (32 * gpu_threads * cpu_id) * sizeof(uint8_t), hipMemcpyHostToDevice),
                     false);

  return true;
}

__host__ HashResult bitcredit_cpu_hash(uint32_t gpu_id, uint32_t cpu_id, uint32_t gpu_threads, uint64_t threads, uint64_t startNounce,
                                       uint32_t expired) {
  uint64_t result[MAX_CPUS];
  uint64_t vcpu[MAX_CPUS];

  const int threadsperblock = 256;

  HashResult r;
  r.nonce = UINT64_MAX;

  memset(result, UINT64_MAX, sizeof(result));
  memset(vcpu, UINT64_MAX, sizeof(vcpu));
  CUDA_CALL_OR_RET_X(hipMemset(d_BitNonce[cpu_id], UINT64_MAX, sizeof(uint64_t)), r);
  CUDA_CALL_OR_RET_X(hipMemset(d_BitVcpu[cpu_id], UINT64_MAX, sizeof(uint64_t)), r);

  dim3 grid(threads / threadsperblock / gpu_threads, gpu_threads);
  dim3 block(threadsperblock);

  bitcredit_gpu_hash<<<grid, block>>>(gpu_threads, cpu_id, threads, (startNounce / gpu_threads), expired, d_BitNonce[cpu_id],
                                      d_BitVcpu[cpu_id]);
  hipDeviceSynchronize();

  CUDA_CALL_OR_RET_X(hipMemcpy(&result[cpu_id], d_BitNonce[cpu_id], sizeof(uint64_t), hipMemcpyDeviceToHost), r);
  CUDA_CALL_OR_RET_X(hipMemcpy(&vcpu[cpu_id], d_BitVcpu[cpu_id], sizeof(uint64_t), hipMemcpyDeviceToHost), r);
  r.nonce = result[cpu_id];
  r.vcpu = vcpu[cpu_id];
  r.cpu_id = cpu_id;
  return r;
}
